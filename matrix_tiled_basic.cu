#include <stdio.h>
#include <hip/hip_runtime.h>
#include <ctime>

#define N  512  
#define TILE_WIDTH 16  

__global__ void matrixMulGPU_Basic(int *a, int *b, int *c) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;

    if (row < N && col < N) {
        for (int k = 0; k < N; ++k) {
            sum += a[row * N + k] * b[k * N + col];
        }
        c[row * N + col] = sum;
    }
}

__global__ void matrixMulGPU_Tiled(int *a, int *b, int *c) {
    int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int col = blockIdx.x * TILE_WIDTH + threadIdx.x;
    int temp = 0;

    __shared__ int tileA[TILE_WIDTH][TILE_WIDTH];
    __shared__ int tileB[TILE_WIDTH][TILE_WIDTH];

    for (int p = 0; p < (N / TILE_WIDTH); ++p) {
        tileA[threadIdx.y][threadIdx.x] = a[row * N + (p * TILE_WIDTH + threadIdx.x)];
        tileB[threadIdx.y][threadIdx.x] = b[(p * TILE_WIDTH + threadIdx.y) * N + col];
        __syncthreads();

        for (int k = 0; k < TILE_WIDTH; ++k) {
            temp += tileA[threadIdx.y][k] * tileB[k][threadIdx.x];
        }
        __syncthreads();
    }
    c[row * N + col] = temp;
}

void matrixMulCPU(int *a, int *b, int *c) {
    for (int row = 0; row < N; ++row) {
        for (int col = 0; col < N; ++col) {
            int sum = 0;
            for (int k = 0; k < N; ++k) {
                sum += a[row * N + k] * b[k * N + col];
            }
            c[row * N + col] = sum;
        }
    }
}

int main() {
    int *a, *b, *c_cpu, *c_gpu_basic, *c_gpu_tiled;
    int size = N * N * sizeof(int);

    hipMallocManaged(&a, size);
    hipMallocManaged(&b, size);
    hipMallocManaged(&c_cpu, size);
    hipMallocManaged(&c_gpu_basic, size);
    hipMallocManaged(&c_gpu_tiled, size);

    for (int row = 0; row < N; ++row) {
        for (int col = 0; col < N; ++col) {
            a[row * N + col] = row;
            b[row * N + col] = col + 2;
            c_cpu[row * N + col] = 0;
            c_gpu_basic[row * N + col] = 0;
            c_gpu_tiled[row * N + col] = 0;
        }
    }

    dim3 threads_per_block(TILE_WIDTH, TILE_WIDTH);
    dim3 num_blocks((N + threads_per_block.x - 1) / threads_per_block.x, 
                    (N + threads_per_block.y - 1) / threads_per_block.y);

    // Time GPU (Basic)
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    matrixMulGPU_Basic<<<num_blocks, threads_per_block>>>(a, b, c_gpu_basic);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("GPU Basic time: %f ms\n", milliseconds);

    // Time GPU (Tiled)
    hipEventRecord(start);
    matrixMulGPU_Tiled<<<num_blocks, threads_per_block>>>(a, b, c_gpu_tiled);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("GPU Tiled time: %f ms\n", milliseconds);

    // Time CPU
    clock_t start_cpu = clock();
    matrixMulCPU(a, b, c_cpu);
    clock_t end_cpu = clock();
    double cpu_time_used = ((double) (end_cpu - start_cpu)) / CLOCKS_PER_SEC * 1000.0; // Convert to milliseconds
    printf("CPU time: %f ms\n", cpu_time_used);

    // Compare results
    for (int i = 0; i < N * N; i++) {
        if (c_cpu[i] != c_gpu_basic[i]) {
            printf("Mismatch found in Basic GPU implementation at index %d\n", i);
            break;
        }
        if (c_cpu[i] != c_gpu_tiled[i]) {
            printf("Mismatch found in Tiled GPU implementation at index %d\n", i);
            break;
        }
    }

    // Free memory
    hipFree(a); hipFree(b);
    hipFree(c_cpu); hipFree(c_gpu_basic); hipFree(c_gpu_tiled);

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
